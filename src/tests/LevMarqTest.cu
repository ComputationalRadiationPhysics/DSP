#include "hip/hip_runtime.h"
#include <iostream>
#include "../LevMarq.h"
#include "../FitFunction.h"
#include <thrust/device_vector.h>
#include "Util.h"

typedef float DATATYPE;
typedef texture<DATATYPE, 2, hipReadModeElementType> tex_t;


__global__ void testFetch(hipTextureObject_t texObj) {
	for(float j = 0; j <= 10; j++) {
		//float i = tex2D<float>(texObj, j, 0.0f);
		float i = getSample(texObj, j, 0);
		printf("Wert = %f", i);
	}
}
__global__ void testKernel(hipTextureObject_t texObj, float* F, float* param) {
	testFetch<<<1,1>>>(texObj);
	/*
	dim3 gs(1,1);
	dim3 bs(10+3,1);
	unsigned int sample_count = 10;
	calcF<Polynom<2> ><<<gs, bs>>>(0, texObj, param, F, 0, sample_count, 1);
	*/
}
int main(int argc, char** argv) {
	const int sample_count = 1000;
	float sample_data[sample_count];
	
	hipArray_t texArray;
	for(int i = 0; i < sample_count; i++) sample_data[i] = i*i+i+1;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipMallocArray(&texArray, &channelDesc, sample_count, 1);
	//hipMalloc((void**)&d_result[i], sizeof(struct fitData) * SAMPLE_COUNT);
	hipMemcpyToArray(texArray, 0, 0, sample_data, sizeof(DATATYPE) * sample_count, hipMemcpyHostToDevice);
	
	// Specify texture
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = texArray;

	// Specify texture object parameters
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0]   = hipAddressModeClamp;
	texDesc.addressMode[1]   = hipAddressModeClamp;
	texDesc.filterMode       = hipFilterModeLinear;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	// Create texture object
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
	hipDeviceSynchronize();
	handleLastError();
	
	hipStream_t stream;
	thrust::device_vector<float> F(13);
	thrust::device_vector<float> param(3);
	hipStreamCreate(&stream);
	//levenbergMarquardt<Polynom<2, 0>, 0>(stream, sample_count, sample_count, 1, 1);
	dim3 gs(1,1);
	dim3 bs(10+3,1);
	//testKernel<<<1,1>>>(texObj, pcast(F), pcast(param));
	//calcF<Polynom<2> ><<<gs, bs, 0, stream>>>(texObj, 0, pcast(param), pcast(F), 0, sample_count, 1);
	levMarqIt<Polynom<2> ><<<1,1>>>(texObj, 10, 10, 0, 1);
	hipDeviceSynchronize();
	handleLastError();
	
	std::cout << "Test done." << std::endl;
	
	hipDestroyTextureObject(texObj);
	hipFreeArray(texArray);
	hipStreamDestroy(stream);
	
	return 0;
}
