#include <iostream>
#include <thrust/device_vector.h>
#include "../LevMarq.h"
#include "../FitFunction.h"
#include "Util.h"

typedef float DATATYPE;
typedef texture<DATATYPE, 2, hipReadModeElementType> tex_t;

int main(int argc, char** argv) {
	const int sample_count = 10;
	float sample_data[sample_count];
	hipArray_t texArray;
	for(int i = 0; i < sample_count; i++) {
		sample_data[i] = i*i+i+1;
		//std::cout << "sample_data[" << i << "] = " << sample_data[i] << std::endl;
	}
	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipMallocArray(&texArray, &channelDesc, sample_count, 1);
	//hipMalloc((void**)&d_result[i], sizeof(struct fitData) * SAMPLE_COUNT);
	hipMemcpyToArray(texArray, 0, 0, sample_data, sizeof(DATATYPE) * sample_count, hipMemcpyHostToDevice);
	
	FitData *fitData;
	hipMalloc((void**)(&fitData), sizeof(FitData));
	
	// Specify texture
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = texArray;

	// Specify texture object parameters
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0]   = hipAddressModeClamp;
	texDesc.addressMode[1]   = hipAddressModeClamp;
	texDesc.filterMode       = hipFilterModeLinear;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	// Create texture object
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
	hipDeviceSynchronize();
	handleLastError();
	
	hipStream_t stream;
	hipStreamCreate(&stream);
	dim3 gs(1,1);
	dim3 bs(sample_count+3,1);
	float* mem;
	const unsigned int window_size = sample_count;
	const size_t SPACE = ((window_size+FitFunction::numberOfParams)*2+(window_size+FitFunction::numberOfParams)*FitFunction::numberOfParams);
	hipMalloc((void**) &mem, sizeof(float)*SPACE);
	levenbergMarquardt<Polynom<2> >(stream, texObj, fitData, sample_count, sample_count, 1, 1, mem);
	hipFree(mem);
	hipDeviceSynchronize();
	handleLastError();
	FitData results[1];
	hipMemcpy(results, fitData, sizeof(results), hipMemcpyDeviceToHost);
	std::cout << results->param[2] << "x²+" << results->param[1] << "x+" << results->param[0]<< std::endl;
	
	std::cout << "Test done." << std::endl;
	hipDestroyTextureObject(texObj);
	hipFreeArray(texArray);
	hipFree(fitData);
	hipStreamDestroy(stream);
	
	return 0;
}
