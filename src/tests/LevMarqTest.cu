#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include "../LevMarq.hpp"
#include "../FitFunction.hpp"
#include "Util.hpp"
#include "Wave.hpp"

typedef float DATATYPE;
typedef texture<DATATYPE, 2, hipReadModeElementType> tex_t;
const unsigned int ORDER = 6;

template <unsigned int order>
float poly(float x) {
	float res;
	for(int i = 0; i <= order; i++) {
		res += std::pow(x,i);
	}
	return res;
}


int main(int argc, char** argv) {
	const int sample_count = 1000;
	float sample_data[sample_count];
	hipArray_t texArray;
	for(int i = 0; i < sample_count; i++) {
		const float x = static_cast<float>(i)/sample_count;
		sample_data[i] = firstWave[i]; //poly<ORDER>(x)
		std::cout << "sample_data[" << i << "] = " << sample_data[i] << std::endl;
	}
	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipMallocArray(&texArray, &channelDesc, sample_count, 1);
	hipMemcpyToArray(texArray, 0, 0, sample_data, sizeof(DATATYPE) * sample_count, hipMemcpyHostToDevice);
	
	FitData *fitData;
	hipMalloc((void**)(&fitData), sizeof(FitData));
	
	// Specify texture
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = texArray;

	// Specify texture object parameters
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0]   = hipAddressModeClamp;
	texDesc.addressMode[1]   = hipAddressModeClamp;
	texDesc.filterMode       = hipFilterModeLinear;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 0;
	
	// Create texture object
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
	hipDeviceSynchronize();
	handleLastError();
	std::cout << "Texture Object created." << std::endl;
	hipStream_t stream;
	hipStreamCreate(&stream);
	float* mem;
	const unsigned int window_size = sample_count;
	const size_t SPACE = ((window_size+Polynom<ORDER>::numberOfParams)*2+(window_size+Polynom<ORDER>::numberOfParams)*Polynom<ORDER>::numberOfParams);
	hipMalloc((void**) &mem, sizeof(float)*SPACE);
	std::cout << "Kernel start." << std::endl;
	//levenbergMarquardt<Polynom<ORDER> >(stream, texObj, fitData, sample_count, sample_count, 1, 1, mem);
	levenbergMarquardt<Gauss>(stream, texObj, fitData, sample_count, sample_count, 1, 1, mem);
	hipFree(mem);
	hipDeviceSynchronize();
	handleLastError();
	FitData results[1];
	hipMemcpy(results, fitData, sizeof(results), hipMemcpyDeviceToHost);
	std::cout << "status=" << results[0].status << std::endl;
	//std::cout << results[0];
	std::cout << results[0].param[0] << "*exp(-((x-" << results[0].param[1] << ")/" << results[0].param[3] << ")**2) + " <<  results[0].param[2] << std::endl;
	std::cout << "Test done." << std::endl;
	hipDestroyTextureObject(texObj);
	hipFreeArray(texArray);
	hipFree(fitData);
	hipStreamDestroy(stream);
	
	return 0;
}
