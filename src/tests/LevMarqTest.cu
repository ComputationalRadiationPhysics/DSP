#include <iostream>
#include "../LevMarq.h"
#include "../FitFunction.h"
typedef float DATATYPE;
typedef texture<DATATYPE, 2, hipReadModeElementType> tex_t;

int main(int argc, char** argv) {
	const int sample_count = 1000;
	float sample_data[sample_count];
	for(int i = 0; i < sample_count; i++) sample_data[i] = (i)*(i)+i+1;
	dataTexture0.filterMode=FILTER_MODE;
	dataTexture0.addressMode[0] = hipAddressModeClamp;
	dataTexture0.addressMode[1] = hipAddressModeClamp;
	hipArray* texArray;

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<DATATYPE>();
	hipMallocArray(&texArray, &channelDesc, sample_count, 1);
	//hipMalloc((void**)&d_result[i], sizeof(struct fitData) * SAMPLE_COUNT);
	hipBindTextureToArray(dataTexture0, texArray);
	hipMemcpyToArray(texArray, 0, 0, sample_data, sizeof(DATATYPE) * sample_count, hipMemcpyHostToDevice);
	
	hipStream_t stream;
	
	hipStreamCreate(&stream);
	levenbergMarquardt<Polynom<2, 0>, 0>(sample_count, 1000, 1, 50);
	
	return 0;
}
