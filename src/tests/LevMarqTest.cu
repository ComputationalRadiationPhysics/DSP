#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include "../LevMarq.h"
#include "../FitFunction.h"
#include "Util.h"

typedef float DATATYPE;
typedef texture<DATATYPE, 2, hipReadModeElementType> tex_t;


__global__ void testFetch(hipTextureObject_t texObj) {
	for(float j = 0; j <= 10; j++) {
		//float i = tex2D<float>(texObj, j, 0.0f);
		float i = getSample(texObj, j, 0);
		printf("Wert = %f", i);
	}
}
__global__ void testKernel(hipTextureObject_t texObj, float* F, float* param) {
	testFetch<<<1,1>>>(texObj);
	/*
	dim3 gs(1,1);
	dim3 bs(10+3,1);
	unsigned int sample_count = 10;
	calcF<Polynom<2> ><<<gs, bs>>>(0, texObj, param, F, 0, sample_count, 1);
	*/
}
int main(int argc, char** argv) {
	const int sample_count = 1000;
	float sample_data[sample_count];
	hipArray_t texArray;
	for(int i = 0; i < sample_count; i++) {
		sample_data[i] = i*i+i+1;
		//std::cout << "sample_data[" << i << "] = " << sample_data[i] << std::endl;
	}
	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	hipMallocArray(&texArray, &channelDesc, sample_count, 1);
	//hipMalloc((void**)&d_result[i], sizeof(struct fitData) * SAMPLE_COUNT);
	hipMemcpyToArray(texArray, 0, 0, sample_data, sizeof(DATATYPE) * sample_count, hipMemcpyHostToDevice);
	
	FitData<3> *fitData;
	hipMalloc((void**)(&fitData), sizeof(FitData<3>));
	
	// Specify texture
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = texArray;

	// Specify texture object parameters
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0]   = hipAddressModeClamp;
	texDesc.addressMode[1]   = hipAddressModeClamp;
	texDesc.filterMode       = hipFilterModeLinear;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 0;

	// Create texture object
	hipTextureObject_t texObj = 0;
	hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);
	hipDeviceSynchronize();
	handleLastError();
	
	hipStream_t stream;
	hipStreamCreate(&stream);
	dim3 gs(1,1);
	dim3 bs(sample_count+3,1);
	levMarqIt<Polynom<2> ><<<1,1>>>(texObj, fitData, sample_count, sample_count, 0, 1);
	hipDeviceSynchronize();
	handleLastError();
	FitData<3> results[1];
	hipMemcpy(results, fitData, sizeof(results), hipMemcpyDeviceToHost);
	std::cout << results->param[2] << "x²+" << results->param[1] << "x+" << results->param[0]<< std::endl;
	
	std::cout << "Test done." << std::endl;
	hipDestroyTextureObject(texObj);
	hipFreeArray(texArray);
	hipFree(fitData);
	hipStreamDestroy(stream);
	
	return 0;
}
