#include "hip/hip_runtime.h"
#include <iostream>
#include <thrust/device_vector.h>
#include "../UtilKernels.h"
#include "Util.h"
#include <cstdio>

typedef int TYPE;
template <class T>
void cpuMatMul(thrust::device_vector<T>& A, thrust::device_vector<T>& B, thrust::device_vector<T>& C, int lcols, int lrows, int rcols, int rrows ) {
	for ( uint32_t i = 0; i < lrows; i++ ) {
        for ( uint32_t j = 0; j < rcols; j++ ) {
            for ( uint32_t k = 0; k < rrows; k++ ) {
                // C[i][j] += A[i][k] * B[k][j]
                C[ i * rcols + j ] += A[ i * lcols + k ] * B[ k * rcols + j ];
            }
        }
    }

}

template <class T>
/* __global__ */ void gpuMatProduct(T* a, T* b, T* c, unsigned int lc, unsigned int lr, unsigned int rc, unsigned int rr) {
	MatrixAccess<TYPE> left(a, lc, lr), right(b, rc, rr), result(c,rc,lr);
	MatMul(result, left, right);
}

int main(int argc, char** argv) {
    srand( ( unsigned ) time( NULL ) );
	hipDeviceReset();
	int w1 = 1025, h1 = 1, w2= 1, h2 = 1025;
	thrust::device_vector<TYPE> A(w1*h1), B(w2*h2), C(h1*w2), D(h1*w2), AT(w1*h1);
	
	random_mat(A,w1,h1);
	
	random_mat(B,w2,h2);
	
	//for(int i = 0; i < w1; i++) A[i] = i;
	//for(int i = 0; i < w1; i++) B[i] = i;
	
	//printMat(A, h1, w1);
	//printMat(B, h2, w2);
	
	std::cout << "CPU Mat Prod" << std::endl;
	cpuMatMul(A,B,D,w1,h1,w2,h2);
	
	std::cout << "GPU Mat Prod" << std::endl;
	gpuMatProduct(pcast(A), pcast(B), pcast(C), w1,h1,w2,h2);
	
	hipDeviceSynchronize();
	handleLastError();
	std::cout << "Test Mat Prod" << std::endl;
	bool passed = true;
	for(int i = 0; i < h1*w2; i++) {
		if(C[i] / D[i] > 1.0001 || C[i] / D[i] < 0.9999) {
			std::cout << "Element " << i << " (" << C[i] << "!=" << D[i] << ") is incorrect!" << std::endl;
			getchar();
			passed = false;
		}
	}
	if(passed) std::cout << "TEST PASSED!" << std::endl;
	/*
	std::cout << "Test Orthogonal Product" << std::endl;
	
	for(int i = 0; i < w; i++) {
		for(int j = 0; j < h; j++) {
			AT[i*h+j] = A[j*w+i];
		}
	}
	
	//cpuMatMul(A,AT,D,50,100,100,50);
	//orthogonalMatProd(pcast(A), pcast(C), 50, 100);
	cpuMatMul(A,AT,D,h,w,w,h);
	
	handleLastError();
	orthogonalMatProd(pcast(A), pcast(C), h, w);
	handleLastError();
	
	std::cout << "Test Mat Prod" << std::endl;
	for(int i = 0; i < h*w; i++) {
		if(C[i]!=D[i]) {
			std::cout << "Element " << i << " (" << C[i] << "!=" << D[i] << ") is incorrect!" << std::endl;
			getchar();
		}
	}
	*/
	//printMat(A, h, w);
	//printMat(AT, w, h);
	//printMat(C, h, h);
	//printMat(D, h, h);
	return 0;
}
