#include "Node.h"
#include "LevMarq.h"
#include <vector>


typedef texture<DATATYPE, 2, hipReadModeElementType> tex_t;

Node::Node(int deviceIdentifier, InputBuffer* input, OutputBuffer* output) :
	deviceIdentifier(deviceIdentifier),
	finish(false),
	iBuffer(input),
	oBuffer(output)
{
	pthread_create(&thread_tid, NULL, run_helper, this); 
}

void Node::run() {
	
	/* 
	 * Example for Texture usage found here
	 * http://www.math.ntu.edu.tw/~wwang/mtxcomp2010/download/cuda_04_ykhung.pdf
	*/
	/* Initialise device */
	hipSetDevice(deviceIdentifier);
	std::cout << "Device " << deviceIdentifier << " initialised." << std::endl;
	
	/* Set texture parameter */
	dataTexture0.filterMode=FILTER_MODE;
	dataTexture0.addressMode[0] = hipAddressModeClamp;
	dataTexture0.addressMode[1] = hipAddressModeClamp;
	dataTexture1.filterMode=FILTER_MODE;
	dataTexture1.addressMode[0] = hipAddressModeClamp;
	dataTexture1.addressMode[1] = hipAddressModeClamp;
	dataTexture2.filterMode=FILTER_MODE;
	dataTexture2.addressMode[0] = hipAddressModeClamp;
	dataTexture2.addressMode[1] = hipAddressModeClamp;
	dataTexture3.filterMode=FILTER_MODE;
	dataTexture3.addressMode[0] = hipAddressModeClamp;
	dataTexture3.addressMode[1] = hipAddressModeClamp;
	dataTexture4.filterMode=FILTER_MODE;
	dataTexture4.addressMode[0] = hipAddressModeClamp;
	dataTexture4.addressMode[1] = hipAddressModeClamp;
	dataTexture5.filterMode=FILTER_MODE;
	dataTexture5.addressMode[0] = hipAddressModeClamp;
	dataTexture5.addressMode[1] = hipAddressModeClamp;
	
	//TODO: REDUCE MAGIC NUMBERS
	std::vector<hipArray*> texArrays;
	std::vector<fitData*> d_result;
	hipStream_t streams[6];
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<DATATYPE>();
	for(int i = 0; i <= 5; i++) {
		/* Allocate memory */
		texArrays.push_back(NULL);
		d_result.push_back(NULL);
		hipStreamCreate(&streams[i]);
		hipMallocArray(&texArrays[i], &channelDesc, SAMPLE_COUNT, CHUNK_COUNT);
		hipMalloc((void**)&d_result[i], sizeof(struct fitData) * SAMPLE_COUNT);	
	}
	hipBindTextureToArray(dataTexture0, texArrays[0]);
	hipBindTextureToArray(dataTexture1, texArrays[1]);
	hipBindTextureToArray(dataTexture2, texArrays[2]);
	hipBindTextureToArray(dataTexture3, texArrays[3]);
	hipBindTextureToArray(dataTexture4, texArrays[4]);
	hipBindTextureToArray(dataTexture5, texArrays[5]);
	int tex = 0;
	while(!iBuffer->isFinished()) {
		fitData result[6][CHUNK_COUNT];
		
		/* Take a chunk from ringbuffer and copy to GPU */
		/* Block ringbuffer */
		Chunk *c = iBuffer->reserveTailTry();
		/* Copy to device */

		if(c != NULL) {
			cudaMemcpyToArrayAsync(texArrays[tex], 0, 0, &c->front(), 
                                   sizeof(DATATYPE) * c->size(), 
                                   hipMemcpyHostToDevice, streams[tex]);
			/* Free ringbuffer 
               This is possible because at the moment we use pageable (non-pinnend)
               host memory for the ringbuffer.
               In this case hipMemcpy...Async will first copy data to a staging 
               buffer and then return. Only copying from staging buffer to final 
               destination is asynchronous.
               Should we switch to pinnend host memory for the ringbuffer we must
               not call iBuffer->freeTail() directly after hipMemcpy..Async.
               See 
http://developer.download.nvidia.com/compute/cuda/4_1/rel/toolkit/docs/online/sync_async.html#MemcpyAsynchronousBehavior
             */
			iBuffer->freeTail();
			std::cout << "Chunk taken from input buffer (device " << deviceIdentifier << "). " << iBuffer->getSize() << " elements remaining in queue." << std::endl;
			hipMemcpy(d_result[tex], result[tex], sizeof(struct fitData) * CHUNK_COUNT, hipMemcpyHostToDevice);
			++tex;
			/* 6 Chucks are copied to the GPU */
			if(tex == 6) {
				tex = 0;
				/* Start kernel */
				kernel<0><<<SAMPLE_COUNT, 1, 0, streams[0]>>>(SAMPLE_COUNT, INTERPOLATION_COUNT, d_result[0]);
				kernel<1><<<SAMPLE_COUNT, 1, 0, streams[1]>>>(SAMPLE_COUNT, INTERPOLATION_COUNT, d_result[1]);
				kernel<2><<<SAMPLE_COUNT, 1, 0, streams[2]>>>(SAMPLE_COUNT, INTERPOLATION_COUNT, d_result[2]);
				kernel<3><<<SAMPLE_COUNT, 1, 0, streams[3]>>>(SAMPLE_COUNT, INTERPOLATION_COUNT, d_result[3]);
				kernel<4><<<SAMPLE_COUNT, 1, 0, streams[4]>>>(SAMPLE_COUNT, INTERPOLATION_COUNT, d_result[4]);
				kernel<5><<<SAMPLE_COUNT, 1, 0, streams[5]>>>(SAMPLE_COUNT, INTERPOLATION_COUNT, d_result[5]);
				/* Get result */
				for(int i = 0; i <= 5; i++) {				
					hipMemcpyAsync(result[i], d_result[i], sizeof(struct fitData) * CHUNK_COUNT, hipMemcpyDeviceToHost, streams[i]);
				}
				for(int i = 0; i <= 5; i++) {									
					/* Sync */
					hipStreamSynchronize(streams[i]);
					/* Push result to output buffer */
					
					for(int j = 0; j < CHUNK_COUNT; j++) {
						if(true) { //TODO: Check for fit quality
							oBuffer->writeFromHost(result[i][j]);
						}
					}				
				}
			}	
		}
	}
	hipUnbindTexture(dataTexture0);
	hipUnbindTexture(dataTexture1);
	hipUnbindTexture(dataTexture2);
	hipUnbindTexture(dataTexture3);
	hipUnbindTexture(dataTexture4);
	hipUnbindTexture(dataTexture5);
	
	for(int i = 0; i <= 5; i++) {
		hipFreeArray(texArrays[i]);
		hipFree(d_result[i]);
		hipStreamDestroy(streams[i]);
	}

	oBuffer->producerQuit();
}
