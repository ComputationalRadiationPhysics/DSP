#include "Node.h"
#include "Constants.h"
#include "Textures.h"
#include "LevMarq.cu" //TODO: Nico erinnern nen Header zu schreiben

Node::Node(int deviceIdentifier, InputBuffer* input, OutputBuffer* output) :
	deviceIdentifier(deviceIdentifier),
	finish(false),
	iBuffer(input),
	oBuffer(output)
{
	pthread_create(&thread_tid, NULL, run_helper, this);
}

int Node::copyChunk(hipArray *texArray, fitData* d_result) {
	
	fitData result[SAMPLE_COUNT];
		
	/* Take a chunk from ringbuffer and copy to GPU */
	/* Block ringbuffer */
	SampleChunk *c = iBuffer->reserveTail();
	/* Copy to device */
	hipMemcpyToArray(texArray, 0, 0, c, sizeof(Precision)*SAMPLE_COUNT*CHUNK_COUNT, hipMemcpyHostToDevice);
	/* Free ringbuffer */
	iBuffer->freeTail();
		hipMemcpy(d_result, result, sizeof(struct fitData) * SAMPLE_COUNT, hipMemcpyHostToDevice);
	/* Start kernel */

	kernel<<<SAMPLE_COUNT, 1>>>(SAMPLE_COUNT, d_result);
	
	/* Get result */
	hipMemcpy(result, d_result, sizeof(struct fitData) * CHUNK_COUNT, hipMemcpyDeviceToHost);
	/* Push result to output buffer */
	//TODO!
	
	return 0;
}
void Node::run() {
	
	/* 
	 * Example for Texture usage found here
	 * http://www.math.ntu.edu.tw/~wwang/mtxcomp2010/download/cuda_04_ykhung.pdf
	 */
	
	/* Initialise device */
	hipSetDevice(deviceIdentifier);
	
	/* Allocate memory */
	hipArray *texArray;
	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<Precision>();
	hipMallocArray(&texArray, &channelDesc, SAMPLE_COUNT, CHUNK_COUNT); 

	/* Set texture parameter */
	dataTexture.filterMode=FILTER_MODE;
	dataTexture.addressMode[0] = hipAddressModeClamp;
	dataTexture.addressMode[1] = hipAddressModeClamp;
	
	/* bind texture to texture reference*/
	hipBindTextureToArray(dataTexture, texArray);
	
	fitData* d_result;
	hipMalloc((void**)&d_result, sizeof(struct fitData) * SAMPLE_COUNT);

	while(!finish) {
		copyChunk(texArray,  d_result);		
	}
	
	/* Empty the the iBuffer */
	while(!iBuffer->isEmpty()) {
		copyChunk(texArray,  d_result);	
	}
	
	hipUnbindTexture(dataTexture);
	hipFreeArray(texArray);
	hipFree(d_result);
}

int Node::stop() {
	/* Called by main thread if all sample data is transfered to the devices */
	finish = true;
	return 0;
}
