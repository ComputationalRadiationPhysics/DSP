#include "Node.h"
#include "Constants.h"
#include "Textures.h"

Node::Node(int deviceIdentifier, InputBuffer* input, OutputBuffer* output) :
	deviceIdentifier(deviceIdentifier),
	finish(false),
	iBuffer(input),
	oBuffer(output)
{
	pthread_create(&thread_tid, NULL, run_helper, this);
}

int Node::copyChunk(hipArray *texArray, fitData* d_result) {
	
	fitData result[CHUNK_COUNT];
		
	/* Take a chunk from ringbuffer and copy to GPU */
	/* Block ringbuffer */
	SampleChunk *c = iBuffer->reserveTail();
	/* Copy to device */
	hipMemcpyToArray(texArray, 0, 0, c, sizeof(Precision)*SAMPLE_COUNT*CHUNK_COUNT, hipMemcpyHostToDevice);
	/* Free ringbuffer */
	iBuffer->freeTail();
	std::cout << "Chunk taken from input bufffer" << std::endl;
	hipMemcpy(d_result, result, sizeof(struct fitData) * CHUNK_COUNT, hipMemcpyHostToDevice);
	/* Start kernel */

	kernel<<<SAMPLE_COUNT, 1>>>(SAMPLE_COUNT, d_result);
	/* Get result */
	hipMemcpy(result, d_result, sizeof(struct fitData) * CHUNK_COUNT, hipMemcpyDeviceToHost);
	/* Push result to output buffer */
	
	for(int i = 0; i < CHUNK_COUNT; i++) {
		if(true) { //TODO: Check for fit quality
			oBuffer->writeFromHost(&result[i]);
		}
	}
	
	
	return 0;
}
void Node::run() {
	
	/* 
	 * Example for Texture usage found here
	 * http://www.math.ntu.edu.tw/~wwang/mtxcomp2010/download/cuda_04_ykhung.pdf
	 */
	
	/* Initialise device */
	hipSetDevice(deviceIdentifier);
	
	/* Allocate memory */
	hipArray *texArray;
	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<Precision>();
	hipMallocArray(&texArray, &channelDesc, SAMPLE_COUNT, CHUNK_COUNT); 

	/* Set texture parameter */
	dataTexture.filterMode=FILTER_MODE;
	dataTexture.addressMode[0] = hipAddressModeClamp;
	dataTexture.addressMode[1] = hipAddressModeClamp;
	
	/* bind texture to texture reference*/
	hipBindTextureToArray(dataTexture, texArray);
	
	fitData* d_result;
	hipMalloc((void**)&d_result, sizeof(struct fitData) * SAMPLE_COUNT);

	while(!finish) {
		copyChunk(texArray,  d_result);		
	}
	
	/* Empty the the iBuffer */
	while(!iBuffer->isEmpty()) {
		copyChunk(texArray,  d_result);	
	}
	
	hipUnbindTexture(dataTexture);
	hipFreeArray(texArray);
	hipFree(d_result);
}

int Node::stop() {
	/* Called by main thread if all sample data is transfered to the devices */
	finish = true;
	return 0;
}
