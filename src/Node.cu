#include "Node.h"
#include "Constants.h"
#include "Textures.h"

Node::Node(int deviceIdentifier, InputBuffer* input) :
	deviceIdentifier(deviceIdentifier),
	finish(false),
	iBuffer(input)
{
	pthread_create(&thread_tid, NULL, run_helper, this);
}

void Node::run() {
	
	/* 
	 * Example for Texture usage found here
	 * http://www.math.ntu.edu.tw/~wwang/mtxcomp2010/download/cuda_04_ykhung.pdf
	 */
	
	/* Initialise device */
	hipSetDevice(deviceIdentifier);
	
	/* Allocate memory */
	hipArray *texArray;
	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<Precision>();
	hipMallocArray(&texArray, &channelDesc, SAMPLE_COUNT, CHUNK_COUNT); 

	/* Set texture parameter */
	sampleData.filterMode=FILTER_MODE;
	sampleData.addressMode[0] = hipAddressModeBorder;
	//sampleData.addressMode[1] = hipAddressModeBorder;
	
	/* bind texture to texture reference*/
	hipBindTextureToArray(sampleData, texArray);

	while(!finish) {
		/* Take a chunk from ringbuffer and copy to GPU */
			/* Block ringbuffer */
			SampleChunk *c = iBuffer->reserveTail();
			/* Copy to device */
			hipMemcpyToArray(texArray, 0, 0, c, sizeof(Precision)*SAMPLE_COUNT*CHUNK_COUNT, hipMemcpyHostToDevice);
			/* Free ringbuffer */
			iBuffer->freeTail(c);
		/* Start kernel */
		
		/* Get result */
		//hipMemcpy(...)
		

		
		/* Push result to output buffer */
		
	}
	
	/* Empty the the iBuffer */
	while(!iBuffer->isEmpty()) {
		/* Take a chunk from ringbuffer and copy to GPU */
			/* Block ringbuffer */
			SampleChunk *c = iBuffer->reserveTail();
			/* Copy to device */
			hipMemcpyToArray(texArray, 0, 0, c, sizeof(Precision)*SAMPLE_COUNT*CHUNK_COUNT, hipMemcpyHostToDevice);
			/* Free ringbuffer */
			iBuffer->freeTail(c);
		/* Start kernel */
		
		/* Get result */
		//hipMemcpy(...)
		

		
		/* Push result to output buffer */
	}
	
	/* Free GPU Memory*/
	hipUnbindTexture(sampleData);
	hipFreeArray(texArray);
}

int Node::stop() {
	/* Called by main thread if all sample data is transfered to the devices */
	finish = true;
	return 0;
}
