#include "Node.h"
#include "LevMarq.h"
#include <vector>
#include "FitFunction.h"

typedef texture<DATATYPE, 2, hipReadModeElementType> tex_t;

Node::Node(int deviceIdentifier, InputBuffer* input, OutputBuffer* output) :
	deviceIdentifier(deviceIdentifier),
	finish(false),
	iBuffer(input),
	oBuffer(output)
{
	pthread_create(&thread_tid, NULL, run_helper, this); 
}

void Node::run() {
	
	/* 
	 * Example for Texture usage found here
	 * http://www.math.ntu.edu.tw/~wwang/mtxcomp2010/download/cuda_04_ykhung.pdf
	*/
	/* Initialise device */
	
	typedef WindowPolynom<2> Fit;
	const unsigned int window_size = 100;//SAMPLE_COUNT/INTERPOLATION_COUNT;
	hipSetDevice(deviceIdentifier);
	
	hipTextureObject_t texObj[numberOfTextures];
	hipStream_t streams[numberOfTextures];
	hipArray_t texArrays[numberOfTextures];
	bool textureEmpty[numberOfTextures];
	
	// Specify texture object parameters
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0]   = hipAddressModeClamp;
	texDesc.addressMode[1]   = hipAddressModeClamp;
	texDesc.filterMode       = hipFilterModeLinear;
	texDesc.readMode         = hipReadModeElementType;
	texDesc.normalizedCoords = 0;
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
	FitData<Fit::numberOfParams> results[CHUNK_COUNT];
	typedef FitData<Fit::numberOfParams> FitDataArray[numberOfTextures][CHUNK_COUNT];
	FitData<Fit::numberOfParams> *fitData;
	hipMalloc((void**)(&fitData), sizeof(FitDataArray));
	#pragma loop unroll
	for(unsigned int i = 0; i < numberOfTextures; i++) {
		hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
		hipMallocArray(&texArrays[i], &channelDesc, SAMPLE_COUNT, CHUNK_COUNT);

		// Specify texture
		hipResourceDesc resDesc;
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypeArray;
		resDesc.res.array.array = texArrays[i];

		// Create texture object
		hipCreateTextureObject(&texObj[i], &resDesc, &texDesc, NULL);
		textureEmpty[i] = true;
	}
	std::cout << "Device " << deviceIdentifier << " initialised." << std::endl;
	int tex = 0;
	unsigned int lastTexture = 0;
	while(!iBuffer->isFinished() || !textureEmpty[lastTexture]) {
		/* copy results back */
		if(!textureEmpty[tex]) {
			hipMemcpyAsync(results, &fitData[tex*CHUNK_COUNT], sizeof(results), hipMemcpyDeviceToHost, streams[tex]);
			hipStreamSynchronize(streams[tex]);
			for(int i = 0; i < CHUNK_COUNT; i++) {
					oBuffer->writeFromHost(results[i]);
			}
			textureEmpty[tex] = true;
		}
		//TODO: Racecondition
		if(!iBuffer->isFinished()) {
			/* Take a chunk from ringbuffer and copy to GPU */
			/* Block ringbuffer */
			Chunk *c = iBuffer->reserveTailTry();
			/* Copy to device */
			if(c != NULL) {
				cudaMemcpyToArrayAsync(texArrays[tex], 0, 0, &c->front(), 
		                               sizeof(DATATYPE) * c->size(), 
		                               hipMemcpyHostToDevice, streams[tex]);
	  			/* Free ringbuffer 
		           This is possible because at the moment we use pageable (non-pinnend)
		           host memory for the ringbuffer.
		           In this case hipMemcpy...Async will first copy data to a staging 
		           buffer and then return. Only copying from staging buffer to final 
		           destination is asynchronous.
		           Should we switch to pinnend host memory for the ringbuffer we must
		           not call iBuffer->freeTail() directly after hipMemcpy..Async.
		           See 
				   http://developer.download.nvidia.com/compute/cuda/4_1/rel/toolkit/docs/online/sync_async.html#MemcpyAsynchronousBehavior
		         */
				iBuffer->freeTail();
				std::cout << "Chunk taken from input buffer (device " << deviceIdentifier << "). " << iBuffer->getSize() << " elements remaining in queue." << std::endl;
				levenbergMarquardt<Fit>(streams[tex], texObj[tex], &fitData[tex*CHUNK_COUNT], SAMPLE_COUNT, window_size, CHUNK_COUNT, INTERPOLATION_COUNT);
				lastTexture = tex;
				tex = (tex+1)%numberOfTextures;
				textureEmpty[tex] = false;
			}
		}
	}
	for(unsigned int i = 0; i < numberOfTextures; i++) {
		hipDestroyTextureObject(texObj[i]);
		hipFreeArray(texArrays[i]);
		hipStreamDestroy(streams[i]);
	}
	hipFree(fitData);
	oBuffer->producerQuit();
}
