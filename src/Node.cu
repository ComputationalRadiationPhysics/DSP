#include "Node.h"
#include "Constants.h"
#include "Textures.h"

Node::Node(int deviceIdentifier, InputBuffer* input) :
	deviceIdentifier(deviceIdentifier),
	finish(false),
	iBuffer(input)
{
	pthread_create(&thread_tid, NULL, run_helper, this);
}

void Node::run() {
	/* Initialise device */
	hipSetDevice(deviceIdentifier);
	
	/* Allocate memory */
	hipArray *texArray;
	
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<Precision>();
	hipMallocArray(&texArray, &channelDesc, SAMPLE_COUNT, CHUNK_COUNT); 

	/* Set texture parameter */
	sampleData.filterMode=FILTER_MODE;
	sampleData.addressMode[0] = hipAddressModeBorder;
	//sampleData.addressMode[1] = hipAddressModeBorder;
	
	/* bind texture to texture reference*/
	hipBindTextureToArray(sampleData, texArray);

	while(!finish) {
		/* Take a chunk from ringbuffer and copy to GPU */
			/* Block ringbuffer */
			SampleChunk *c = iBuffer->reserveTail();
			/* Copy to device */
			hipMemcpyToArray(texArray, 0, 0, c, sizeof(Precision)*SAMPLE_COUNT*CHUNK_COUNT, hipMemcpyHostToDevice);
			/* Free ringbuffer */
			iBuffer->freeTail(c);
		/* Start kernel */
		
		/* Get result */
		//hipMemcpy(...)
		

		
		/* Push result to output buffer */
		
	}
	
	/* Free GPU Memory*/
	hipUnbindTexture(sampleData);
	hipFreeArray(texArray);
}

int Node::stop() {
	/* Called by main thread if all sample data is transfered to the devices */
	finish = true;
	return 0;
}
