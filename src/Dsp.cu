#include <vector>
#include <iostream>

#include "Node.hpp"
#include "OutputStream.hpp"
#include "Constants.hpp"
#include "DataReader.hpp"
#include "StopWatch.hpp"


//Taken from https://github.com/ComputationalRadiationPhysics/HaseOnGpu
//Author: Erik Zenker, Carlchristian Eckert
std::vector<unsigned> getFreeDevices(unsigned maxGpus){
	hipDeviceProp_t prop;
	int minMajor = MIN_COMPUTE_CAPABILITY_MAJOR;
	int minMinor = MIN_COMPUTE_CAPABILITY_MINOR;
	int count;
	std::vector<unsigned> devices;

	// Get number of devices
	hipGetDeviceCount(&count);

	// Check devices for compute capability and if device is busy
	unsigned devicesAllocated = 0;
	for(int i=0; i < count; ++i){
		hipGetDeviceProperties(&prop, i);
		if( (prop.major > minMajor) || (prop.major == minMajor && prop.minor >= minMinor) ){
			hipSetDevice(i);
			int* occupy; //TODO: occupy gets allocated, but never hipFree'd -> small memory leak!
			if(hipMalloc((void**) &occupy, sizeof(int)) == hipSuccess){
				devices.push_back(i);
				devicesAllocated++;
				if(devicesAllocated == maxGpus)
					break;
			}
		}
	}
	// Exit if no device was found
	if(devices.size() == 0){
		std::cout << "None of the free CUDA-capable devices is sufficient!" << std::endl;
		exit(1);
	}

	// Print device information
	hipSetDevice(devices.at(0));
	std::cout << "Found " << int(devices.size()) << " available CUDA devices with Compute Capability >= " << minMajor << "." << minMinor << "):" << std::endl;
	for(unsigned i=0; i<devices.size(); ++i){
		hipGetDeviceProperties(&prop, devices[i]);
		std::cout << "[" << devices[i] << "] " << prop.name << " (Compute Capability " << prop.major << "." << prop.minor << ")" << std::endl;
	}

	return devices;

}
  
int main(int argc, char* argv[]) {
	
	/* Get number of devices */
	int numberOfDevices;
	hipError_t err;
	err = hipGetDeviceCount(&numberOfDevices);
	std::vector<unsigned> freeDevices = getFreeDevices(maxNumberOfDevices);
	
	/* Check the cuda runtime environment */
	if(err != hipSuccess) {
		std::cerr << "Something went wrong during the creation the context, or no Cuda capable devices are installed on the system." << std::endl;
		std::cerr << "Exit." << std::endl;
		return 1;
	}

	std::string input_filename = FILENAME_TESTFILE;
	std::string output_filename =  OUTPUT_FILENAME;

	if(argc > 1) {
		input_filename = argv[1];	
	}
	if(argc > 2) {
		output_filename = argv[2];
	}
	
	std::cout << "Args read (" << input_filename << ", " << output_filename << ")" << std::endl;

    int nSample = -1;
    int nSegments = -1;
    int nWaveforms = -1;

    DataReader::readHeader(input_filename, nSample, nSegments, nWaveforms);
	std::cout << "Header read. File compatible." << std::endl;

	/* Initialize input buffer (with dynamic elements) */
    Chunk dc(CHUNK_COUNT * nSample);
    std::fill(dc.begin(), dc.end(), 0);
	InputBuffer inputBuffer(CHUNK_BUFFER_COUNT, 1, dc);
    /* Initialize output buffer (with static elements) */
	OutputStream os(output_filename, freeDevices.size());
	
	std::cout << "Buffer created." << std::endl;
	
    DataReader reader(input_filename, &inputBuffer, CHUNK_COUNT);
    std::cout << "DataReader created." << std::endl;

	std::vector<Node*> devices;
	StopWatch sw;
	sw.start();
	for(int i = 0; i < freeDevices.size(); i++) {
		/* Start threads to handle Nodes */
		devices.push_back(new Node(freeDevices[i], &inputBuffer, os.getBuffer()));
	}
	reader.readToBuffer();
	std::cout << "Data read." << std::endl;
	
	std::cout << "Nodes created." << std::endl;
		

	//Make sure all results are written back
	os.join();
	sw.stop();
	std::cout << "Time: " << sw << std::endl;
	//std::cout << "Throuput: " << 382/(sw.elapsedSeconds()) << "MiB/s."<< std::endl;
	return 0;
}
