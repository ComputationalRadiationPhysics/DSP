#include <vector>
#include <iostream>

#include "Node.h"

int main(int argc, char* argv[]) {
	
	/* Get number of devices */
	int numberOfDevices;
	hipError_t err;
	err = hipGetDeviceCount(&numberOfDevices);
	
	/* Check the cuda runtime environment */
	if(err != hipSuccess) {
		std::cerr << "Something went wrong during the creation the context, or no Cuda capable devices are installed on the system." << std::endl;
		std::cerr << "Exit." << std::endl;
		return 1;
	}
	
	/* Initialise input buffer */
	InputBuffer inputBuffer;
	
	
	
	std::vector<Node> devices;
	for(int i = 0; i < numberOfDevices; i++) {
		/* Start threads to handle Nodes */
		devices.push_back(Node(i, &inputBuffer));
	}
	
	return 0;
}
