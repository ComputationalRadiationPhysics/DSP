#include <vector>
#include <iostream>

#include "Node.h"
#include "Constants.h"

int main(int argc, char* argv[]) {
	
	/* Get number of devices */
	int numberOfDevices;
	hipError_t err;
	err = hipGetDeviceCount(&numberOfDevices);
	
	/* Check the cuda runtime environment */
	if(err != hipSuccess) {
		std::cerr << "Something went wrong during the creation the context, or no Cuda capable devices are installed on the system." << std::endl;
		std::cerr << "Exit." << std::endl;
		return 1;
	}
	
	/* Initialise input buffer */
	InputBuffer inputBuffer(CHUNK_BUFFER_COUNT);
	
	
	
	std::vector<Node> devices;
	for(int i = 0; i < numberOfDevices; i++) {
		/* Start threads to handle Nodes */
		devices.push_back(Node(i, &inputBuffer));
	}
	
	return 0;
}
